#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#define NX 256
#define NY 256
#define NZ 256
#define BLOCK_SIZE 16

__global__ void poissonKernel(double *phi, double *rhs, int nx, int ny, int nz) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx < nx && idy < ny && idz < nz) {
        double dx = 1.0 / (nx - 1);
        double dy = 1.0 / (ny - 1);
        double dz = 1.0 / (nz - 1);

        double laplacian = 0.0;
        laplacian += (phi[(idx + 1) * ny * nz + idy * nz + idz] - 2.0 * phi[idx * ny * nz + idy * nz + idz] + phi[(idx - 1) * ny * nz + idy * nz + idz]) / (dx * dx);
        laplacian += (phi[idx * ny * nz + (idy + 1) * nz + idz] - 2.0 * phi[idx * ny * nz + idy * nz + idz] + phi[idx * ny * nz + (idy - 1) * nz + idz]) / (dy * dy);
        laplacian += (phi[idx * ny * nz + idy * nz + (idz + 1)] - 2.0 * phi[idx * ny * nz + idy * nz + idz] + phi[idx * ny * nz + idy * nz + (idz - 1)]) / (dz * dz);

        phi[idx * ny * nz + idy * nz + idz] = phi[idx * ny * nz + idy * nz + idz] - 0.5 * laplacian / rhs[idx * ny * nz + idy * nz + idz];
    }
}

int main() {
    int nx = NX;
    int ny = NY;
    int nz = NZ;

    double *phi_host, *rhs_host;
    hipHostMalloc((void **)&phi_host, nx * ny * nz * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void **)&rhs_host, nx * ny * nz * sizeof(double), hipHostMallocDefault);

    // Initialize phi and rhs arrays
    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                phi_host[i * ny * nz + j * nz + k] = 0.0;
                rhs_host[i * ny * nz + j * nz + k] = 1.0;
            }
        }
    }

    double *phi_dev, *rhs_dev;
    hipMalloc((void **)&phi_dev, nx * ny * nz * sizeof(double));
    hipMalloc((void **)&rhs_dev, nx * ny * nz * sizeof(double));

    hipMemcpy(phi_dev, phi_host, nx * ny * nz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rhs_dev, rhs_host, nx * ny * nz * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, (nz + block.z - 1) / block.z);

    for (int iter = 0; iter < 100; iter++) {
        poissonKernel<<<grid, block>>>(phi_dev, rhs_dev, nx, ny, nz);
        hipDeviceSynchronize();
    }

    hipMemcpy(phi_host, phi_dev, nx * ny * nz * sizeof(double), hipMemcpyDeviceToHost);

    // Output the result
    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                phi_host[i * ny * nz + j * nz + k];
                // std::cout << "phi[" << i << "," << j << "," << k << "] = " << phi_host[i * ny * nz + j * nz + k] << std::endl;
            }
        }
    }

    hipFree(phi_dev);
    hipFree(rhs_dev);
    hipHostFree(phi_host);
    hipHostFree(rhs_host);

    return 0;
}